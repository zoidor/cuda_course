#include "hip/hip_runtime.h"
//Udacity HW 6
//Poisson Blending

/* Background
   ==========

   The goal for this assignment is to take one image (the source) and
   paste it into another image (the destination) attempting to match the
   two images so that the pasting is non-obvious. This is
   known as a "seamless clone".

   The basic ideas are as follows:

   1) Figure out the interior and border of the source image
   2) Use the values of the border pixels in the destination image 
      as boundary conditions for solving a Poisson equation that tells
      us how to blend the images.
   
      No pixels from the destination except pixels on the border
      are used to compute the match.

   Solving the Poisson Equation
   ============================

   There are multiple ways to solve this equation - we choose an iterative
   method - specifically the Jacobi method. Iterative methods start with
   a guess of the solution and then iterate to try and improve the guess
   until it stops changing.  If the problem was well-suited for the method
   then it will stop and where it stops will be the solution.

   The Jacobi method is the simplest iterative method and converges slowly - 
   that is we need a lot of iterations to get to the answer, but it is the
   easiest method to write.

   Jacobi Iterations
   =================

   Our initial guess is going to be the source image itself.  This is a pretty
   good guess for what the blended image will look like and it means that
   we won't have to do as many iterations compared to if we had started far
   from the final solution.

   ImageGuess_prev (Floating point)
   ImageGuess_next (Floating point)

   DestinationImg
   SourceImg

   Follow these steps to implement one iteration:

   1) For every pixel p in the interior, compute two sums over the four neighboring pixels:
      Sum1: If the neighbor is in the interior then += ImageGuess_prev[neighbor]
             else if the neighbor in on the border then += DestinationImg[neighbor]

      Sum2: += SourceImg[p] - SourceImg[neighbor]   (for all four neighbors)

   2) Calculate the new pixel value:
      float newVal= (Sum1 + Sum2) / 4.f  <------ Notice that the result is FLOATING POINT
      ImageGuess_next[p] = min(255, max(0, newVal)); //clamp to [0, 255]


    In this assignment we will do 800 iterations.
   */



#include "utils.h"
#include <thrust/host_vector.h>

template<typename OType, typename IType, typename OperatorType>
__global__ void map(OType * out, const  IType * in, const size_t sz, OperatorType op)
{
	size_t pos = blockDim.x * blockIdx.x + threadIdx.x;
	if(pos >= sz) return;
	out[pos] = op(in[pos]);
}

void generate_mask(unsigned char * mask, const uchar4 * img, const size_t sz)
{
	const size_t num_threads = 128;
	size_t num_blocks = (size_t)std::ceil(sz / (double)num_threads);
	if(num_blocks == 0)  num_blocks = 1;
	
	auto op =  []__device__(const uchar4 el) -> unsigned char
	{
		if(el.x == el.y && el.y == el.z && el.z == 255) return 255;
		return 0;
	};
	map<<<num_blocks, num_threads>>>(mask, img, sz, op);
}   

void your_blend(const uchar4* const h_sourceImg,  //IN
                const size_t numRowsSource, const size_t numColsSource,
                const uchar4* const h_destImg, //IN
                uchar4* const h_blendedImg) //OUT
{

	uchar4 *d_sourceImg = NULL;
	uchar4 *d_destImg = NULL;

	checkCudaErrors(hipMalloc(&d_sourceImg, sizeof(uchar4) * numRowsSource * numColsSource));
	checkCudaErrors(hipMalloc(&d_destImg, sizeof(uchar4) * numRowsSource * numColsSource));


  /* To Recap here are the steps you need to implement
  
     1) Compute a mask of the pixels from the source image to be copied
        The pixels that shouldn't be copied are completely white, they
        have R=255, G=255, B=255.  Any other pixels SHOULD be copied.

	*/

	unsigned char *d_mask = NULL;
	checkCudaErrors(hipMalloc(&d_mask, sizeof(unsigned char) * numRowsSource * numColsSource));
	generate_mask(d_mask, d_sourceImg, numRowsSource * numColsSource);
	
/*     
	2) Compute the interior and border regions of the mask.  An interior
        pixel has all 4 neighbors also inside the mask.  A border pixel is
        in the mask itself, but has at least one neighbor that isn't.

     3) Separate out the incoming image into three separate channels

     4) Create two float(!) buffers for each color channel that will
        act as our guesses.  Initialize them to the respective color
        channel of the source image since that will act as our intial guess.

     5) For each color channel perform the Jacobi iteration described 
        above 800 times.

     6) Create the output image by replacing all the interior pixels
        in the destination image with the result of the Jacobi iterations.
        Just cast the floating point values to unsigned chars since we have
        already made sure to clamp them to the correct range.

      Since this is final assignment we provide little boilerplate code to
      help you.  Notice that all the input/output pointers are HOST pointers.

      You will have to allocate all of your own GPU memory and perform your own
      memcopies to get data in and out of the GPU memory.

      Remember to wrap all of your calls with checkCudaErrors() to catch any
      thing that might go wrong.  After each kernel call do:

      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

      to catch any errors that happened while executing the kernel.
  */

	hipFree(d_sourceImg);
	hipFree(d_mask);
	hipFree(d_destImg);
}
