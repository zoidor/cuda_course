#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

//TODO: investigate wether the kernels can be reduced. Investigate whether block size can be used to improve performance. 
template<typename device_scan_operator>
__global__ void cuda_scan_in_block(const size_t * d_in, size_t * d_out, size_t * d_out_tails, const size_t length_vec, device_scan_operator op, unsigned int identity)
{
	extern __shared__ size_t s_block_scan_mem[];
	size_t * s_block_scan1 = s_block_scan_mem;
	size_t * s_block_scan2 = s_block_scan_mem + blockDim.x;
	const int tid = threadIdx.x;
	const int pos = tid + blockDim.x * blockIdx.x;

	if(pos >= length_vec) return;

	s_block_scan1[tid] = pos == 0 ? identity : d_in[pos -1];
	__syncthreads();

	for(int shift = 1; shift <= blockDim.x; shift *= 2)
	{
		const int prev = tid - shift;
		if(prev >= 0)
			s_block_scan2[tid] = op(s_block_scan1[tid], s_block_scan1[prev]);
		else
			s_block_scan2[tid] = s_block_scan1[tid];
		__syncthreads();
		size_t * tmp = s_block_scan1;
		s_block_scan1 = s_block_scan2;
		s_block_scan2 = tmp;
	} 

	d_out[pos] = s_block_scan1[tid];

	if(tid == blockDim.x - 1) 
	{
		d_out_tails[blockIdx.x] =  s_block_scan1[tid];
	}
}

template<typename device_scan_operator>
__global__ void cuda_scan_post_process(const size_t * in_vec, const size_t * in_vec_tails, size_t * out_vec, const size_t length_vec, device_scan_operator op, unsigned int cuda_scan_in_block_b_size, const size_t start)
{
	const size_t pos = blockDim.x * blockIdx.x + threadIdx.x;

	if(pos >= length_vec) 
	{
		return;
	}

	//we want to allow the previous and current block sizes to be different
	const int idx_of_block_in_scan = pos / cuda_scan_in_block_b_size;
	size_t el = op(in_vec[pos], in_vec_tails[idx_of_block_in_scan]);
	out_vec[pos] = op(el, start);
}

void print(const size_t * d_vec, const size_t length)
{
	std::vector<size_t> toP(length);
	hipMemcpy(toP.data(), d_vec, sizeof(size_t) * length, hipMemcpyDeviceToHost);
	for(auto el : toP)	
	{
		printf("%i ", (int)el);
	}
	printf("\n--------------- %i *------\n", (int)length);
}

template<typename operatorType>
void scan(size_t * const d_vec, const size_t length, unsigned int identity_element, operatorType op, unsigned int start_element){
	const int K = 1024;
	const int K2 = 1024;
	const int num_blocks = std::max(1, static_cast<int>(std::ceil(length / (double)K)));
	const int num_blocks2 = std::max(1, static_cast<int>(std::ceil(length / (double)K2)));
	size_t * d_out_vec = NULL;
	size_t * d_out_vec_tails = NULL;

	checkCudaErrors(hipMalloc(&d_out_vec, sizeof(size_t) * length));
	checkCudaErrors(hipMalloc(&d_out_vec_tails, sizeof(size_t) * num_blocks));
	cuda_scan_in_block<<<num_blocks, K, sizeof(size_t) * K * 2>>>(d_vec, d_out_vec, d_out_vec_tails, length, op, identity_element);
	checkCudaErrors(hipGetLastError());

	if(num_blocks == 1)
	{

		hipMemcpy(d_vec, d_out_vec, length * sizeof(size_t), hipMemcpyDeviceToDevice);
	}
	else
	{
		scan(d_out_vec_tails, num_blocks, identity_element, op, identity_element);
		cuda_scan_post_process<<<num_blocks2, K2>>>(d_out_vec, d_out_vec_tails, d_vec, length, op, K, start_element);
	}

	checkCudaErrors(hipGetLastError());		

	checkCudaErrors(hipFree(d_out_vec));	
	checkCudaErrors(hipFree(d_out_vec_tails));	
}


template<typename OpType>
__global__ void cuda_get_flags(const unsigned int * vals, size_t * flags, size_t numElems, OpType op)
{
	const size_t pos = blockDim.x * blockIdx.x + threadIdx.x;
	if(pos >= numElems) return;
	
	unsigned int is_bit_active = op(vals[pos]);
	flags[pos] = is_bit_active;
}


__global__ void scatter(const size_t * scatter_0, const size_t * scatter_1, const size_t * flags_0, const unsigned int * in, unsigned int * out, const size_t length)
{
	const size_t pos = threadIdx.x + blockDim.x * blockIdx.x;
	if(pos >= length) return;

	size_t scatter_pos = flags_0[pos] ? scatter_0[pos] : scatter_1[pos];
	
	if(scatter_pos >= length) return;

	out[scatter_pos] = in[pos]; 
}  


void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               size_t numElems)
{ 

	//numElems = 32;
 	int numbits = sizeof(unsigned int) * 8;
	
	unsigned int * vals1 = NULL;
	unsigned int * vals2 = NULL;


	unsigned int * pos1 = NULL;
	unsigned int * pos2 = NULL;

	

	size_t * scatter_loc0 = NULL;
	size_t * scatter_loc1 = NULL;
	size_t * flags = NULL;

	checkCudaErrors(hipMalloc(&scatter_loc0, sizeof(size_t) * numElems));
	checkCudaErrors(hipMalloc(&flags, sizeof(size_t) * numElems));
	checkCudaErrors(hipMalloc(&scatter_loc1, sizeof(size_t) * numElems));


	checkCudaErrors(hipMalloc(&vals1, sizeof(unsigned int) * numElems));
	checkCudaErrors(hipMalloc(&vals2, sizeof(unsigned int) * numElems));
	checkCudaErrors(hipMalloc(&pos1, sizeof(unsigned int) * numElems));
	checkCudaErrors(hipMalloc(&pos2, sizeof(unsigned int) * numElems));
	
	checkCudaErrors(hipMemcpy(vals1, d_inputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(pos1, d_inputPos, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));

	const int K = 512;
	const int num_blocks = (int)ceil(numElems / (double)K);

	auto scan_op = [] __device__ (size_t el1, size_t el2) -> size_t {return el1 + el2;};


	for(int i = 0; i < numbits; ++i)
	{	unsigned int mask = pow(2, i);
		auto mask_op_0 = [mask]__device__ (unsigned int el) -> unsigned int
			  {
				return (unsigned int)((el & mask) == 0);
			  };

		auto mask_op_1 = [mask]__device__ (unsigned int el) -> unsigned int
			  {
				return (unsigned int)((el & mask) != 0);
			  }; 

		
		cuda_get_flags<<<num_blocks, K>>>(vals1, scatter_loc0, numElems, mask_op_0);
		checkCudaErrors(hipGetLastError());		
		
		checkCudaErrors(hipMemcpy(flags, scatter_loc0, sizeof(size_t) * numElems, hipMemcpyDeviceToDevice));
		
		scan(scatter_loc0, numElems, 0, scan_op, 0);
		checkCudaErrors(hipGetLastError());		
		

		size_t start1;
		checkCudaErrors(hipMemcpy(&start1, &scatter_loc0[numElems - 1], sizeof(size_t), hipMemcpyDeviceToHost));

		size_t is_last_1;		
		checkCudaErrors(hipMemcpy(&is_last_1, &flags[numElems - 1], sizeof(size_t), hipMemcpyDeviceToHost));
		
		start1 += is_last_1;
		cuda_get_flags<<<num_blocks, K>>>(vals1, scatter_loc1, numElems, mask_op_1);
		scan(scatter_loc1, numElems, 0, scan_op, start1);
		checkCudaErrors(hipGetLastError());		
				
		scatter<<<num_blocks, K>>>(scatter_loc0, scatter_loc1, flags, vals1, vals2, numElems);
		scatter<<<num_blocks, K>>>(scatter_loc0, scatter_loc1, flags, pos1, pos2, numElems);

		std::swap(vals1, vals2);
		std::swap(pos1, pos2);
	}

	checkCudaErrors(hipFree(scatter_loc0));
	checkCudaErrors(hipFree(scatter_loc1));
	checkCudaErrors(hipFree(flags));

	checkCudaErrors(hipMemcpy(d_outputVals, vals1, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(d_outputPos, pos1, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));

	checkCudaErrors(hipFree(vals1));
	checkCudaErrors(hipFree(vals2));
	checkCudaErrors(hipFree(pos1));
	checkCudaErrors(hipFree(pos2));
}
